
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

__global__ void sync_conv_groups() { }

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, vector<Blob<Dtype>*>* top) {
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = (*top)[i]->mutable_gpu_data();
    const Dtype* weight = this->blobs_[0]->gpu_data();

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      const Dtype alpha = 1.0;
      const Dtype beta = 0.0;

      cudnnConvolutionFwdAlgo_t algo;

      // get the desired convolution algorithm
      CUDNN_CHECK(cudnnGetConvolutionForwardAlgorithm(handle_[g],
        bottom_descs_[i],
        filter_desc_,
        conv_descs_[i],
        top_descs_[i],
        CUDNN_CONVOLUTION_FWD_NO_WORKSPACE,
        0,  // memoryLimitInBytes,
        &algo));

      // get minimum size of the workspace needed for the desired algorithm
      size_t workspaceSizeInBytes;

      CUDNN_CHECK(cudnnGetConvolutionForwardWorkspaceSize(handle_[g],
        bottom_descs_[i],
        filter_desc_,
        conv_descs_[i],
        top_descs_[i],
        algo,
        &workspaceSizeInBytes));

      void *workspace = NULL;

      // Filters.
      CUDNN_CHECK(cudnnConvolutionForward(handle_[g],
            std::reinterpret_cast<void *>(&alpha),
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            filter_desc_, weight + weight_offset_ * g,
            conv_descs_[i],
            algo, workspace, workspaceSizeInBytes,
            std::reinterpret_cast<void *>(&beta),
            top_descs_[i], top_data + top_offset_ * g));

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
        Dtype alpha = 1.0;
        Dtype beta = 1.0;
        CUDNN_CHECK(cudnnAddTensor(handle_[g], CUDNN_ADD_SAME_C,
              std::reinterpret_cast<void *>(&alpha),
              bias_desc_, bias_data + bias_offset_ * g,
              std::reinterpret_cast<void *>(&beta),
              top_descs_[i], top_data + top_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups<<<1, 1>>>();
  }
}

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
    caffe_gpu_set(this->blobs_[0]->count(), Dtype(0), weight_diff);
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
    caffe_gpu_set(this->blobs_[1]->count(), Dtype(0), bias_diff);
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        const Dtype alpha = 1.0;
        const Dtype beta = 1.0;
        CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0*this->group_ + g],
              std::reinterpret_cast<void *>(&alpha),
              top_descs_[i],  top_diff + top_offset_ * g,
              std::reinterpret_cast<void *>(&beta),
              bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        const Dtype alpha = 1.0;
        const Dtype beta = 1.0;
        const Dtype* bottom_data = (*bottom)[i]->gpu_data();
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(handle_[1*this->group_ + g],
              std::reinterpret_cast<void *>(&alpha),
              bottom_descs_[i], bottom_data + bottom_offset_ * g,
              top_descs_[i],    top_diff + top_offset_ * g,
              conv_descs_[i], std::reinterpret_cast<void *>(&beta),
              filter_desc_, weight_diff + weight_offset_ * g));
      }

      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        const Dtype alpha = 1.0;
        const Dtype beta = 0.0;
        Dtype* bottom_diff = (*bottom)[i]->mutable_gpu_diff();
        CUDNN_CHECK(cudnnConvolutionBackwardData(handle_[2*this->group_ + g],
              std::reinterpret_cast<void *>(&alpha),
              filter_desc_, weight + weight_offset_ * g,
              top_descs_[i],    top_diff + top_offset_ * g,
              conv_descs_[i], std::reinterpret_cast<void *>(&beta),
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups<<<1, 1>>>();
  }
}

INSTANTIATE_CLASS(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
